#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include "tempo.h"

#define TAM 30800
#define L 10
#define C 10

//aloca memória global- RAM e CPU
__managed__ int m1[C][L];
__managed__ char genoma0[TAM];
__managed__ char genoma1[TAM];
__managed__ char genoma2[TAM];
__managed__ char genoma3[TAM];
__managed__ char genoma4[TAM];
__managed__ char genoma5[TAM];
__managed__ char genoma6[TAM];
__managed__ char genoma7[TAM];
__managed__ char genoma8[TAM];
__managed__ char genoma9[TAM];
__managed__ int i = 0;

//funcao compara
//===============================================================
__global__ void compara_genoma(char genomaa[TAM], char genomab[TAM], int w, int z, int *qtd_iguais, int *qtd_dif)

{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (genomaa[idx] == '0' && genomab[idx] == '0')
    {
        // break;
    }
    else
    {

        if (genomaa[idx] == genomab[idx])
        { //compara os dois nucleotideos

            atomicAdd(qtd_iguais, 1);
        }
        else
        {

            atomicAdd(qtd_dif, 1);
        }
    }

    m1[z][w] = *qtd_iguais;
}

void mostra_genomas()
{
    int i = 0;

    for (i = 0; i < TAM; i++)
    {

        printf("\n | %5d  | %c | %c | %c | %c | %c | %c | %c | %c | %c | %c | ", i,
               genoma0[i], genoma1[i], genoma2[i], genoma3[i], genoma4[i], genoma5[i], genoma6[i],
               genoma7[i], genoma8[i], genoma9[i]);
    }
}

void inicializa_vetor()
{
    for (int z = 0; z <= TAM; z++)
    {
        genoma0[z] = '0';
        genoma1[z] = '0';
        genoma2[z] = '0';
        genoma3[z] = '0';
        genoma4[z] = '0';
        genoma5[z] = '0';
        genoma6[z] = '0';
        genoma7[z] = '0';
        genoma8[z] = '0';
        genoma9[z] = '0';
    }
}

void inicializa_matriz()
{
    int z, v;
    printf("\n inicializando matrizes ...\n");
    for (z = 0; z < L; z++)
        for (v = 0; v < C; v++)
        {
            m1[z][v] = 0;
        }
}

void mostra_resultado()
{
    int z, v;
    printf("\n");
    printf("\n  RESULTADO COMPARAÇÃO DE GENOMAS\n\n");
    for (v = 0; v < C; v++)
    {
        printf("   %d   ", v);
    }
    printf("\n");
    for (z = 0; z < L; z++)
    {
        printf("%d", z);
        for (v = 0; v < C; v++)
        {

            printf("  %5.2d", m1[z][v]);
        }
        printf("\n");
    }
    printf("\n\n\n");
}

void carrega_genoma()
{

    char c;

    //genoma 1
    FILE *file1;

    //abre arquivo para leitura
    file1 = fopen("genomas/AY278741.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file1)) != EOF)
    { //le um nucleotideo do primeiro genoma enquanto não chegor ao final do arquivo

        genoma0[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file1);

    //-----------------------------
    //genoma 2
    FILE *file2;

    //abre arquivo para leitura
    file2 = fopen("genomas/KY417146.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;

    while ((c = getc(file2)) != EOF)
    {

        genoma1[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file2);

    //-----------------------------
    //genoma 3
    FILE *file3;

    //abre arquivo para leitura
    file3 = fopen("genomas/MK211376.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;

    while ((c = getc(file3)) != EOF)
    {

        genoma2[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file3);

    //-----------------------------
    //genoma 4
    FILE *file4;

    //abre arquivo para leitura
    file4 = fopen("genomas/MN908947.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file4)) != EOF)
    {

        genoma3[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file4);

    //-----------------------------
    //genoma 5
    FILE *file5;

    //abre arquivo para leitura
    file5 = fopen("genomas/MN996532.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file5)) != EOF)
    {

        genoma4[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file5);

    //-----------------------------
    //genoma 6
    FILE *file6;

    //abre arquivo para leitura
    file6 = fopen("genomas/USA-AZ1-2020.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file6)) != EOF)
    {

        genoma5[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file6);

    //-----------------------------
    //genoma 7
    FILE *file7;

    //abre arquivo para leitura
    file7 = fopen("genomas/USA-CA1-2020.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file7)) != EOF)
    {

        genoma6[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file7);

    //-----------------------------
    //genoma 8
    FILE *file8;

    //abre arquivo para leitura
    file8 = fopen("genomas/USA-IL1-2020.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file8)) != EOF)
    {

        genoma7[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file8);

    //-----------------------------
    //genoma 9
    FILE *file9;

    //abre arquivo para leitura
    file9 = fopen("genomas/USA-TX1-2020.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file9)) != EOF)
    {

        genoma8[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file9);

    //-----------------------------
    //genoma 9
    FILE *file10;

    //abre arquivo para leitura
    file10 = fopen("genomas/WHUHAN-WH04-2020.txt", "r");

    //carrega o conteudo do arquivo no vetor genoma1
    i = 0;
    while ((c = getc(file9)) != EOF)
    {

        genoma9[i] = c;
        i++;
    }

    //fecha arquivo
    fclose(file10);
}

int main(int argc, char *argv[0])
{

    tempo1();
    //cria var do tipo ponteiro
    int *num_iguais;
    int *num_dif;

    //aloca memória na ram e cpu
    hipMallocManaged(&num_iguais, 4);
    hipMallocManaged(&num_dif, 4);

    //inicializa os ponteiros
    *num_iguais = 0;
    *num_dif = 0;

    //inicializa os vetores e a matriz de resultado
    inicializa_vetor();
    inicializa_matriz();

    //carrega os genomas
    carrega_genoma();

    //mostra os genomas
    //mostra_genomas();

    //array de array dos genomas
    char *array_gen[10] = {genoma0, genoma1, genoma2, genoma3, genoma4, genoma5, genoma6, genoma7, genoma8, genoma9};

    //executa a função de comparação
    for (int k = 0; k < 9; k++)
    {
        for (int j = k + 1; j <= 9; j++)
        {
            //printf("\n");
            //printf("Comparação executada na thread  %d \n", omp_get_thread_num());
            compara_genoma<<<700, 44>>>(array_gen[k], array_gen[j], k, j, num_iguais, num_dif);
            hipDeviceSynchronize();
            *num_iguais = 0;
            *num_dif=0;
        }
    }

    mostra_resultado();

    tempo2();
    printf("\n O tempo total de execução foi de");
    tempoFinal("mili segundos", argv[0], MSGLOG);
}